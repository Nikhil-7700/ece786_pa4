#include <stdio.h>
#include <fstream>
#include <iostream>
#include <vector>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void matrix_multiply(const float *input, float *output, const float *Umatrix, int size, int qubit) 
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int mask = 1 << qubit;
  int index = x ^ mask;
  if (x > (size - mask)) return;
  
  if((x/mask) % 2 != 1) {
   output[x] = Umatrix[0] * input[x] + Umatrix[1] * input[index];
   output[index] = Umatrix[2] * input[x] + Umatrix[3] * input[index];
  } 
}

int main(int argc, char *argv[])
{
    char *trace_file; // Variable that holds trace file name;
    trace_file = argv[1];
    
    // read input matrix and vector from file
    ifstream file(trace_file);

    // Read the 2x2 matrix
    float matrix_1[4];
    for (int i = 0; i < 4; i++) 
    {
        file >> matrix_1[i];
    }

    float matrix_2[4];
    for (int i = 0; i < 4; i++) 
    {
        file >> matrix_2[i];
    }

    float matrix_3[4];
    for (int i = 0; i < 4; i++) 
    {
        file >> matrix_3[i];
    }
    
    float matrix_4[4];
    for (int i = 0; i < 4; i++) 
    {
        file >> matrix_4[i];
    }
    
    float matrix_5[4];
    for (int i = 0; i < 4; i++) 
    {
        file >> matrix_5[i];
    }
    
    float matrix_6[4];
    for (int i = 0; i < 4; i++) 
    {
        file >> matrix_6[i];
    }

    // Read the input vector
    vector<float> quantum_state;
    float value;
    while (file >> value) 
    {
      quantum_state.push_back(value);
    }

    // Extract 6 target qubit indices from the end of the input
	float target_qubits[6];
	for (int i = 5; i >= 0; --i) {
        target_qubits[i] = quantum_state.back();
        quantum_state.pop_back();
    }

    // Size of input vector
    int n;
    n = quantum_state.size();

    // Compute size of input vector and matrix
    size_t size = n * sizeof(float);
    size_t size_m = 4 * sizeof(float);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input matrix U
    float *h_U = (float *)malloc(size_m);

    // Allocate the host output vector B
    float *h_B = (float *)malloc(size);

    // Initialize the host input vector and matrix
    for (int i = 0; i < n; ++i)
    {
        h_A[i] = quantum_state[i];
    }

    for (int i = 0; i < 4; i++) 
    {
        h_U[i]= matrix_1[i];
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);
    // Allocate the device input matrix U
    float *d_U = NULL;
    hipMalloc((void **)&d_U, size_m);
    // Allocate the device output vector B
    float *d_B = NULL;
    hipMalloc((void **)&d_B, size);

    // Copy the host input A and U in host memory to the device input vectors in device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_U, h_U, size_m, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 64;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;

    //Timing Report
    //struct timeval begin, end; 
    //gettimeofday (&begin, NULL);

    //Applying Qubit Gate 1
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_U, n, quantum_state[0]);
    hipDeviceSynchronize();
    hipMemcpy(h_A, d_B, size, hipMemcpyDeviceToHost);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    for (int i = 0; i < 4; i++) 
    {
        h_U[i]= matrix_2[i];
    }
    hipMemcpy(d_U, h_U, size_m, hipMemcpyHostToDevice);
    //Applying Qubit Gate 2
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_U, n, quantum_state[1]);
    hipDeviceSynchronize();
    hipMemcpy(h_A, d_B, size, hipMemcpyDeviceToHost);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    for (int i = 0; i < 4; i++) 
    {
        h_U[i]= matrix_3[i];
    }
    hipMemcpy(d_U, h_U, size_m, hipMemcpyHostToDevice);
    //Applying Qubit Gate 3
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_U, n, quantum_state[2]);
    hipDeviceSynchronize();
    hipMemcpy(h_A, d_B, size, hipMemcpyDeviceToHost);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    for (int i = 0; i < 4; i++) 
    {
        h_U[i]= matrix_4[i];
    }
    hipMemcpy(d_U, h_U, size_m, hipMemcpyHostToDevice);
    //Applying Qubit Gate 4
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_U, n, quantum_state[3]);
    hipDeviceSynchronize();
    hipMemcpy(h_A, d_B, size, hipMemcpyDeviceToHost);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    for (int i = 0; i < 4; i++) 
    {
        h_U[i]= matrix_5[i];
    }
    hipMemcpy(d_U, h_U, size_m, hipMemcpyHostToDevice);
    //Applying Qubit Gate 5
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_U, n, quantum_state[4]);
    hipDeviceSynchronize();
    hipMemcpy(h_A, d_B, size, hipMemcpyDeviceToHost);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    for (int i = 0; i < 4; i++) 
    {
        h_U[i]= matrix_6[i];
    }
    hipMemcpy(d_U, h_U, size_m, hipMemcpyHostToDevice);
    //Applying Qubit Gate 6
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_U, n, quantum_state[5]);
    hipDeviceSynchronize();
    //gettimeofday (&end, NULL); 
    //int time_in_us = 1e6 * (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec);

    // Copy the device result vector in device memory to the host result vector in host memory.
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_U, d_U, size_m, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i)
    {
        printf("%.3f\n", h_B[i]);
    }

    //cout<<"Time in use = "<< time_in_us <<endl;
    
    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_U);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_U);

    // Reset the device and exit
    hipDeviceReset();
    return 0;
}